#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__f_df_btcusdt(
	float * S,
	//
	float * y, float * dy,
	float * w,
	//
	uint * ts__d,
	//
	uint I, uint T, uint L, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x; 
	//uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//uint  i = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (_y < (L*N)) {
		float s = 0;
		FOR(0, i, I) {
			FOR(0, _t, GRAND_T) {
				FOR(0, mega_t, MEGA_T) {
					uint ty        = t_MODE(_t, mega_t);
					uint t_btcusdt = ts__d[_t] + 1 + mega_t;
					assert(t_btcusdt < T);
					//
					float __y = y[ty*I*L*N + i*L*N + _y];
					float __w = w[ i*T*L*N + t_btcusdt*L*N + _y];
					//printf("i=%i _y=%i pos=%i %f %f\n", i, _y, i*L*N + _y, __y, __w);
					if (__y != __y) {
						printf("i=%i _y=%i pos=%i %f %f\n", i, _y, i*L*N + _y, __y, __w);
						assert(0);
					}
					assert(__y >= -100 && __y <= +100);
					//
					float coef = (float)(GRAND_T * MEGA_T * (I*L*N));
					s       += ( score_p2(__y, __w, 2)) / coef * (_y==0 ? 5:1);
					float ds = (dscore_p2(__y, __w, 2)) / coef * (_y==0 ? 5:1);
					//
					//atomicAdd(&dy[ty*I*N*L + i*L*N + _y], ds);
					dy[ty*I*N*L + i*L*N + _y] = ds;
				}
			}
		}
		//
		//atomicAdd(&S[0], s);
		S[_y] = s;
	}
};

float f_df_btcusdt(BTCUSDT_t * btcusdt, float * y__d, float * dy__d, uint * ts__d) {
	uint I=btcusdt->I;
	uint L=btcusdt->L;
	uint N=btcusdt->N;
	uint T=btcusdt->T;
	//
	float * S__d = cudalloc<float>(L*N);
	k__f_df_btcusdt<<<dim3(KERD((L*N), 32)/*,KERD(GRAND_T, 8), *//*KERD(I,4)*/), dim3(32/*,8,*//*4*/)>>>(
		S__d,
		y__d, dy__d,
		btcusdt->serie__d,
		ts__d,
		I, T, L, N
	);
	ATTENDRE_CUDA();
	//
	//
	float * S = gpu_vers_cpu<float>(S__d, L*N);
	float score = 0;
	FOR(0, i, L*N) score += S[i];
	//
	cudafree<float>(S__d);
	    free       (S   );
	//
	return score;
};
#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../../impl_template/tmpl_etc.cu"

static __global__ void k__pourcent_btcusdt_stricte(
	float * inconnue_____somme,
	float *   connue_____somme,
	//
	float * y, float * w,
	uint * ts__d,
	//
	uint I, uint T, uint L, uint N)
{
	uint _y = threadIdx.x + blockIdx.x * blockDim.x;
	uint _t = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint n = _y%N;
	uint l = (_y-n)/L;
	//
	if (n!=0 && _y < L*N && _t < GRAND_T) {
		FOR(0, i, I) {
			FOR(0, mega_t, MEGA_T) {
				uint ty        = t_MODE(_t, mega_t);
				uint t_btcusdt = ts__d[_t] + PLUS_DECALAGE + mega_t;
				//
				uint wpos = i*T*L*N + t_btcusdt*L*N + _y;
				uint ypos = ty*I*L*N + i*L*N + n*L + l;
				//
				float delat_w = w[wpos] / w[wpos - L] - 1;
				float delta_y = y[ypos] / y[ypos - L] - 1;
				//
				float a_t_il_predit = 1*(float)(sng(delat_w) == sng(delta_y));
				//
				if (n == N-1) atomicAdd(&inconnue_____somme[i], a_t_il_predit);
				else          atomicAdd(&  connue_____somme[i], a_t_il_predit);
			}
		}
	}
};

float* pourcent_btcusdt(BTCUSDT_t * btcusdt, float * y__d, uint * ts__d) {
	uint I=btcusdt->I;
	uint L=btcusdt->L;
	uint N=btcusdt->N;
	uint T=btcusdt->T;
	//
	float * inconnue_____somme__d = cudalloc<float>(btcusdt->I);
	float *   connue_____somme__d = cudalloc<float>(btcusdt->I);
	//
	k__pourcent_btcusdt_stricte<<<dim3(KERD((L*N), 16), KERD(GRAND_T, 16)), dim3(16,16)>>>(
		inconnue_____somme__d,
		  connue_____somme__d,
		y__d, btcusdt->serie__d,
		ts__d,
		btcusdt->I, btcusdt->T, btcusdt->L, btcusdt->N
	);
	ATTENDRE_CUDA();
	//
	float * inconnue_____somme = gpu_vers_cpu<float>(inconnue_____somme__d, btcusdt->I);
	float *   connue_____somme = gpu_vers_cpu<float>(  connue_____somme__d, btcusdt->I);
	//
	float * ret = alloc<float>(btcusdt->I * 2);
	FOR(0, i, btcusdt->I) {
		inconnue_____somme[i] = inconnue_____somme[i] / (float)(GRAND_T*MEGA_T*(L      ));
		  connue_____somme[i] =   connue_____somme[i] / (float)(GRAND_T*MEGA_T*(L*N-L-L));
		//
		ret[0*btcusdt->I + i] = inconnue_____somme[i];
		ret[1*btcusdt->I + i] =   connue_____somme[i];
	}
	//
	cudafree<float>(inconnue_____somme__d);
	cudafree<float>(  connue_____somme__d);
	    free       (inconnue_____somme   );
	    free       (  connue_____somme   );
	//
	return ret;
};
